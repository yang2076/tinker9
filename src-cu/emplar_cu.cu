#include "hip/hip_runtime.h"
#include "add.h"
#include "emplar.h"
#include "empole.h"
#include "empole_self.h"
#include "epolar.h"
#include "epolar_trq.h"
#include "glob.spatial.h"
#include "image.h"
#include "launch.h"
#include "md.h"
#include "pme.h"
#include "seq_damp.h"
#include "seq_triangle.h"
#include "switch.h"
#include "tool/gpu_card.h"


namespace tinker {
// Rt Q = G
__device__
void rotQI2GVector(const real (&restrict rot)[3][3], real3 qif,
                   real3& restrict glf)
{
   glf = make_real3(dot3(rot[0][0], rot[1][0], rot[2][0], qif),
                    dot3(rot[0][1], rot[1][1], rot[2][1], qif),
                    dot3(rot[0][2], rot[1][2], rot[2][2], qif));
}


// R G = Q
__device__
void rotG2QIVector(const real (&restrict rot)[3][3], real3 glf,
                   real3& restrict qif)
{
   qif = make_real3(dot3(rot[0][0], rot[0][1], rot[0][2], glf),
                    dot3(rot[1][0], rot[1][1], rot[1][2], glf),
                    dot3(rot[2][0], rot[2][1], rot[2][2], glf));
}


// R G Rt = Q
__device__
void rotG2QIMat_v1(const real (&restrict rot)[3][3], //
                   real glxx, real glxy, real glxz,  //
                   real glyy, real glyz, real glzz,  //
                   real& restrict qixx, real& restrict qixy,
                   real& restrict qixz, real& restrict qiyy,
                   real& restrict qiyz, real& restrict qizz)
{
   real gl[3][3] = {{glxx, glxy, glxz}, {glxy, glyy, glyz}, {glxz, glyz, glzz}};
   real out[3][3] = {{0, 0, 0}, {0, 0, 0}, {0, 0, 0}};
   // out[i][j] = sum(k,m) R[i][k] gl[k][m] Rt[m][j]
   //           = sum(k,m) R[i][k] gl[k][m] R[j][m]
   for (int i = 0; i < 2; ++i)
      for (int j = i; j < 3; ++j)
         for (int k = 0; k < 3; ++k)
            for (int m = 0; m < 3; ++m)
               out[i][j] += rot[i][k] * gl[k][m] * rot[j][m];
   qixx = out[0][0];
   qixy = out[0][1];
   qixz = out[0][2];
   qiyy = out[1][1];
   qiyz = out[1][2];
   // qizz = out[2][2];
   qizz = -(out[0][0] + out[1][1]);
}


// R G Rt = Q
__device__
void rotG2QIMat_v2(const real (&restrict r)[3][3],  //
                   real glxx, real glxy, real glxz, //
                   real glyy, real glyz, real glzz, //
                   real& restrict qixx, real& restrict qixy,
                   real& restrict qixz, real& restrict qiyy,
                   real& restrict qiyz, real& restrict qizz)
{
   // clang-format off
   qixx=r[0][0]*(r[0][0]*glxx+2*r[0][1]*glxy) + r[0][1]*(r[0][1]*glyy+2*r[0][2]*glyz) + r[0][2]*(r[0][2]*glzz+2*r[0][0]*glxz);
   qiyy=r[1][0]*(r[1][0]*glxx+2*r[1][1]*glxy) + r[1][1]*(r[1][1]*glyy+2*r[1][2]*glyz) + r[1][2]*(r[1][2]*glzz+2*r[1][0]*glxz);
   qixy=r[0][0]*(r[1][0]*glxx+r[1][1]*glxy+r[1][2]*glxz) + r[0][1]*(r[1][0]*glxy+r[1][1]*glyy+r[1][2]*glyz) + r[0][2]*(r[1][0]*glxz+r[1][1]*glyz+r[1][2]*glzz);
   qixz=r[0][0]*(r[2][0]*glxx+r[2][1]*glxy+r[2][2]*glxz) + r[0][1]*(r[2][0]*glxy+r[2][1]*glyy+r[2][2]*glyz) + r[0][2]*(r[2][0]*glxz+r[2][1]*glyz+r[2][2]*glzz);
   qiyz=r[1][0]*(r[2][0]*glxx+r[2][1]*glxy+r[2][2]*glxz) + r[1][1]*(r[2][0]*glxy+r[2][1]*glyy+r[2][2]*glyz) + r[1][2]*(r[2][0]*glxz+r[2][1]*glyz+r[2][2]*glzz);
   // clang-format on
   qizz = -(qixx + qiyy);
}


#define rotG2QIMatrix rotG2QIMat_v2


template <class Ver, class ETYP>
__device__
void pair_mplar(                                                          //
   real r2, real3 dR, real mscale, real dscale, real pscale, real uscale, //
   real ci, real3 Id, real Iqxx, real Iqxy, real Iqxz, real Iqyy, real Iqyz,
   real Iqzz, real3 Iud, real3 Iup, real pdi, real pti, //
   real ck, real3 Kd, real Kqxx, real Kqxy, real Kqxz, real Kqyy, real Kqyz,
   real Kqzz, real3 Kud, real3 Kup, real pdk, real ptk, //
   real f, real aewald,                                 //
   real& restrict frcxi, real& restrict frcyi, real& restrict frczi,
   real& restrict frcxk, real& restrict frcyk, real& restrict frczk,
   real& restrict trqxi, real& restrict trqyi, real& restrict trqzi,
   real& restrict trqxk, real& restrict trqyk, real& restrict trqzk,
   real& restrict eo, real& restrict voxx, real& restrict voxy,
   real& restrict voxz, real& restrict voyy, real& restrict voyz,
   real& restrict vozz)
{
   constexpr bool do_e = Ver::e;
   constexpr bool do_g = Ver::g;
   constexpr bool do_v = Ver::v;


   // a rotation matrix that rotates (xr,yr,zr) to (0,0,r); R G = Q
   real rot[3][3];
   real bn[6];
   real sr3, sr5, sr7, sr9;
   real r = REAL_SQRT(r2);
   real invr1 = REAL_RECIP(r);
   {
      real rr2 = invr1 * invr1;
      real rr1 = invr1;
      real rr3 = rr1 * rr2;
      real rr5 = 3 * rr3 * rr2;
      real rr7 = 5 * rr5 * rr2;
      real rr9 = 7 * rr7 * rr2;
      real rr11;
      if CONSTEXPR (do_g) {
         rr11 = 9 * rr9 * rr2;
      }


      if CONSTEXPR (eq<ETYP, EWALD>()) {
         if CONSTEXPR (!do_g) {
            damp_ewald<5>(bn, r, invr1, rr2, aewald);
         } else {
            damp_ewald<6>(bn, r, invr1, rr2, aewald);
         }
      } else if CONSTEXPR (eq<ETYP, NON_EWALD>()) {
         bn[0] = rr1;
         bn[1] = rr3;
         bn[2] = rr5;
         bn[3] = rr7;
         bn[4] = rr9;
         if CONSTEXPR (do_g) {
            bn[5] = rr11;
         }
      }


      // if use_thole
      real ex3, ex5, ex7, ex9;
      damp_thole4(r, pdi, pti, pdk, ptk, ex3, ex5, ex7, ex9);
      sr3 = bn[1] - ex3 * rr3;
      sr5 = bn[2] - ex5 * rr5;
      sr7 = bn[3] - ex7 * rr7;
      sr9 = bn[4] - ex9 * rr9;
      // end if use_thole


      real3 rotz = invr1 * dR;
      // pick a random vector as rotx; rotx and rotz cannot be parallel
      real3 rotx = rotz;
      if (dR.y != 0 || dR.z != 0)
         rotx.x += 1;
      else
         rotx.y += 1;
      // Gram–Schmidt process for rotx with respect to rotz
      rotx -= dot3(rotx, rotz) * rotz;
      // normalize rotx
      real invxlen = REAL_RSQRT(dot3(rotx, rotx));
      rotx = invxlen * rotx;
      real3 roty = cross(rotz, rotx);
      rot[0][0] = rotx.x;
      rot[0][1] = rotx.y;
      rot[0][2] = rotx.z;
      rot[1][0] = roty.x;
      rot[1][1] = roty.y;
      rot[1][2] = roty.z;
      rot[2][0] = rotz.x;
      rot[2][1] = rotz.y;
      rot[2][2] = rotz.z;
   }


   real3 di, dk;
   rotG2QIVector(rot, Id, di);
   rotG2QIVector(rot, Kd, dk);
   real qixx, qixy, qixz, qiyy, qiyz, qizz;
   real qkxx, qkxy, qkxz, qkyy, qkyz, qkzz;
   rotG2QIMatrix(rot, Iqxx, Iqxy, Iqxz, Iqyy, Iqyz, Iqzz, qixx, qixy, qixz,
                 qiyy, qiyz, qizz);
   rotG2QIMatrix(rot, Kqxx, Kqxy, Kqxz, Kqyy, Kqyz, Kqzz, qkxx, qkxy, qkxz,
                 qkyy, qkyz, qkzz);
   real3 uid, uip;
   rotG2QIVector(rot, Iud, uid);
   rotG2QIVector(rot, Iup, uip);
   real3 ukd, ukp;
   rotG2QIVector(rot, Kud, ukd);
   rotG2QIVector(rot, Kup, ukp);


   // phi,dphi/d(x,y,z),d2phi/dd(xx,yy,zz,xy,xz,yz)
   //   0        1 2 3            4  5  6  7  8  9
   real phi1[10] = {0};
   real phi2[10] = {0};
   real phi1z[10] = {0};


   if CONSTEXPR (eq<ETYP, EWALD>()) {
      mscale = 1;
      dscale = 0.5f;
      pscale = 0.5f;
      uscale = 0.5f;
   } else {
      dscale *= 0.5f;
      pscale *= 0.5f;
      uscale *= 0.5f;
   }


   // C-C
   {
      real coef1 = bn[0];
      real coef3 = bn[1] * r;
      // phi_c c
      phi1[0] += coef1 * ck;
      phi2[0] += coef1 * ci;
      phi1z[0] += coef3 * ck;
   }


   // D-C and C-D
   {
      real coef3 = bn[1] * r;
      real coef5 = (bn[1] - bn[2] * r2);
      // phi_d c
      phi1[0] += -coef3 * dk.z;
      phi2[0] += coef3 * di.z;
      phi1z[0] += coef5 * dk.z;
      // dphi_c d
      // phi1[1]; phi1[2];
      phi1[3] += coef3 * ck;
      // phi2[1]; phi2[2];
      phi2[3] += -coef3 * ci;
      // phi1z[1]; phi1z[2];
      phi1z[3] += -coef5 * ck;
   }


   // D-D
   {
      real coef3 = bn[1];
      real coef5 = (bn[1] - bn[2] * r2);
      real coez5 = bn[2] * r;
      real coez7 = (3 * bn[2] - bn[3] * r2) * r;
      // dphi_d d
      phi1[1] += coef3 * dk.x;
      phi1[2] += coef3 * dk.y;
      phi1[3] += coef5 * dk.z;
      phi2[1] += coef3 * di.x;
      phi2[2] += coef3 * di.y;
      phi2[3] += coef5 * di.z;
      phi1z[1] += coez5 * dk.x;
      phi1z[2] += coez5 * dk.y;
      phi1z[3] += coez7 * dk.z;
   }


   // Q-C and C-Q
   {
      real coef3 = bn[1];
      real coef5 = bn[2] * r2;
      real coez5 = bn[2] * r;
      real coez7 = bn[3] * r2 * r;
      // phi_q c
      phi1[0] += coef5 * qkzz;
      phi2[0] += coef5 * qizz;
      phi1z[0] += -(2 * coez5 - coez7) * qkzz;
      // d2phi_c q
      phi1[4] += -coef3 * ck;
      phi1[5] += -coef3 * ck;
      phi1[6] += -(coef3 - coef5) * ck;
      // phi1[7]; phi1[8]; phi1[9];
      phi2[4] += -coef3 * ci;
      phi2[5] += -coef3 * ci;
      phi2[6] += -(coef3 - coef5) * ci;
      // phi2[7]; phi2[8]; phi2[9];
      phi1z[4] += -coez5 * ck;
      phi1z[5] += -coez5 * ck;
      phi1z[6] += -(3 * coez5 - coez7) * ck;
      // phi1z[7]; phi1z[8]; phi1z[9];
   }


   // Q-D and D-Q
   {
      real coef5 = bn[2] * r;
      real coef7 = bn[3] * r2 * r;
      real coez7 = (bn[2] - bn[3] * r2);
      real coez9 = (3 * bn[3] - bn[4] * r2) * r2;
      // dphi_q d
      phi1[1] += -2 * coef5 * qkxz;
      phi1[2] += -2 * coef5 * qkyz;
      phi1[3] += -(2 * coef5 - coef7) * qkzz;
      phi2[1] += 2 * coef5 * qixz;
      phi2[2] += 2 * coef5 * qiyz;
      phi2[3] += (2 * coef5 - coef7) * qizz;
      phi1z[1] += 2 * coez7 * qkxz;
      phi1z[2] += 2 * coez7 * qkyz;
      phi1z[3] += (2 * coez7 - coez9) * qkzz;
      // d2phi_d q
      phi1[4] += coef5 * dk.z;
      phi1[5] += coef5 * dk.z;
      phi1[6] += (3 * coef5 - coef7) * dk.z;
      // phi1[7];
      phi1[8] += 2 * coef5 * dk.x;
      phi1[9] += 2 * coef5 * dk.y;
      //
      phi2[4] += -coef5 * di.z;
      phi2[5] += -coef5 * di.z;
      phi2[6] += -(3 * coef5 - coef7) * di.z;
      // phi2[7];
      phi2[8] += -2 * coef5 * di.x;
      phi2[9] += -2 * coef5 * di.y;
      //
      phi1z[4] += -coez7 * dk.z;
      phi1z[5] += -coez7 * dk.z;
      phi1z[6] += -(3 * coez7 - coez9) * dk.z;
      // phi1z[7];
      phi1z[8] += -2 * coez7 * dk.x;
      phi1z[9] += -2 * coez7 * dk.y;
   }


   // Q-Q
   {
      // d2phi_q q
      real coef5 = bn[2];
      real coef7 = bn[3] * r2;
      real coef9 = bn[4] * r2 * r2;
      real coez7 = bn[3] * r;
      real coez9 = bn[4] * r2 * r;
      real coez11 = bn[5] * r2 * r2 * r;
      //
      phi1[4] += 2 * coef5 * qkxx - coef7 * qkzz;
      phi1[5] += 2 * coef5 * qkyy - coef7 * qkzz;
      phi1[6] += (2 * coef5 - 5 * coef7 + coef9) * qkzz;
      phi1[7] += 4 * coef5 * qkxy;
      phi1[8] += 4 * (coef5 - coef7) * qkxz;
      phi1[9] += 4 * (coef5 - coef7) * qkyz;
      //
      phi2[4] += 2 * coef5 * qixx - coef7 * qizz;
      phi2[5] += 2 * coef5 * qiyy - coef7 * qizz;
      phi2[6] += (2 * coef5 - 5 * coef7 + coef9) * qizz;
      phi2[7] += 4 * coef5 * qixy;
      phi2[8] += 4 * (coef5 - coef7) * qixz;
      phi2[9] += 4 * (coef5 - coef7) * qiyz;
      //
      phi1z[4] += 2 * coez7 * qkxx + (2 * coez7 - coez9) * qkzz;
      phi1z[5] += 2 * coez7 * qkyy + (2 * coez7 - coez9) * qkzz;
      phi1z[6] += (12 * coez7 - 9 * coez9 + coez11) * qkzz;
      phi1z[7] += 4 * coez7 * qkxy;
      phi1z[8] += 4 * (3 * coez7 - coez9) * qkxz;
      phi1z[9] += 4 * (3 * coez7 - coez9) * qkyz;
   }


   #pragma unroll
   for (int i = 0; i < 10; ++i) {
      phi1[i] *= mscale;
      phi2[i] *= mscale;
      phi1z[i] *= mscale;
   }


   if CONSTEXPR (do_e) {
      real e = phi1[0] * ci + phi1[1] * di.x + phi1[2] * di.y + phi1[3] * di.z +
         phi1[4] * qixx + phi1[5] * qiyy + phi1[6] * qizz + phi1[7] * qixy +
         phi1[8] * qixz + phi1[9] * qiyz;
      eo = f * e;
   }


   real phi1d[3] = {0};
   real phi2d[3] = {0};
   real phi1dz[3] = {0};


   // U-C and C-U
   {
      real coe3 = sr3 * r;
      real coe5 = sr3 - sr5 * r2;
      real coed3 = dscale * coe3;
      real coed5 = dscale * coe5;
      real coep3 = pscale * coe3;
      real coep5 = pscale * coe5;
      // phi_u c
      phi1[0] += -(coed3 * ukp.z + coep3 * ukd.z);
      phi2[0] += coed3 * uip.z + coep3 * uid.z;
      phi1z[0] += coed5 * ukp.z + coep5 * ukd.z;
      // dphi_c u
      phi1d[2] += coe3 * ck;
      phi2d[2] += -coe3 * ci;
      phi1dz[2] += -coe5 * ck;
   }


   // U-D and D-U
   {
      real coe3 = sr3;
      real coe5 = sr5 * r2;
      real coez5 = sr5 * r;
      real coez7 = sr7 * r2 * r;
      real coed3 = dscale * coe3;
      real coed5 = dscale * coe5;
      real coedz5 = dscale * coez5;
      real coedz7 = dscale * coez7;
      real coep3 = pscale * coe3;
      real coep5 = pscale * coe5;
      real coepz5 = pscale * coez5;
      real coepz7 = pscale * coez7;
      // dphi_u d
      phi1[1] += coed3 * ukp.x + coep3 * ukd.x;
      phi1[2] += coed3 * ukp.y + coep3 * ukd.y;
      phi1[3] += (coed3 - coed5) * ukp.z + (coep3 - coep5) * ukd.z;
      phi2[1] += coed3 * uip.x + coep3 * uid.x;
      phi2[2] += coed3 * uip.y + coep3 * uid.y;
      phi2[3] += (coed3 - coed5) * uip.z + (coep3 - coep5) * uid.z;
      phi1z[1] += coedz5 * ukp.x + coepz5 * ukd.x;
      phi1z[2] += coedz5 * ukp.y + coepz5 * ukd.y;
      phi1z[3] += (3 * coedz5 - coedz7) * ukp.z + (3 * coepz5 - coepz7) * ukd.z;
      // dphi_d u
      phi1d[0] += coe3 * dk.x;
      phi1d[1] += coe3 * dk.y;
      phi1d[2] += (coe3 - coe5) * dk.z;
      phi2d[0] += coe3 * di.x;
      phi2d[1] += coe3 * di.y;
      phi2d[2] += (coe3 - coe5) * di.z;
      phi1dz[0] += coez5 * dk.x;
      phi1dz[1] += coez5 * dk.y;
      phi1dz[2] += (3 * coez5 - coez7) * dk.z;
   }


   // U-Q and Q-U
   {
      real coe5 = sr5 * r;
      real coe7 = sr7 * r2 * r;
      real coez7 = sr5 - sr7 * r2;
      real coez9 = (3 * sr7 - sr9 * r2) * r2;
      real coed5 = dscale * coe5;
      real coed7 = dscale * coe7;
      real coedz7 = dscale * coez7;
      real coedz9 = dscale * coez9;
      real coep5 = pscale * coe5;
      real coep7 = pscale * coe7;
      real coepz7 = pscale * coez7;
      real coepz9 = pscale * coez9;
      // d2phi_u q
      phi1[4] += coed5 * ukp.z + coep5 * ukd.z;
      phi1[5] += coed5 * ukp.z + coep5 * ukd.z;
      phi1[6] += (3 * coed5 - coed7) * ukp.z + (3 * coep5 - coep7) * ukd.z;
      // phi1[7];
      phi1[8] += 2 * (coed5 * ukp.x + coep5 * ukd.x);
      phi1[9] += 2 * (coed5 * ukp.y + coep5 * ukd.y);
      //
      phi2[4] += -(coed5 * uip.z + coep5 * uid.z);
      phi2[5] += -(coed5 * uip.z + coep5 * uid.z);
      phi2[6] += -(3 * coed5 - coed7) * uip.z - (3 * coep5 - coep7) * uid.z;
      // phi2[7];
      phi2[8] += -2 * (coed5 * uip.x + coep5 * uid.x);
      phi2[9] += -2 * (coed5 * uip.y + coep5 * uid.y);
      //
      phi1z[4] += -(coedz7 * ukp.z + coepz7 * ukd.z);
      phi1z[5] += -(coedz7 * ukp.z + coepz7 * ukd.z);
      phi1z[6] +=
         -(3 * coedz7 - coedz9) * ukp.z - (3 * coepz7 - coepz9) * ukd.z;
      // phi1z[7];
      phi1z[8] += -2 * (coedz7 * ukp.x + coepz7 * ukd.x);
      phi1z[9] += -2 * (coedz7 * ukp.y + coepz7 * ukd.y);
      // dphi_q u
      phi1d[0] += -2 * coe5 * qkxz;
      phi1d[1] += -2 * coe5 * qkyz;
      phi1d[2] += -(2 * coe5 - coe7) * qkzz;
      phi2d[0] += 2 * coe5 * qixz;
      phi2d[1] += 2 * coe5 * qiyz;
      phi2d[2] += (2 * coe5 - coe7) * qizz;
      phi1dz[0] += 2 * coez7 * qkxz;
      phi1dz[1] += 2 * coez7 * qkyz;
      phi1dz[2] += (2 * coez7 - coez9) * qkzz;
   }


   real3 frc, trq1, trq2;
   if CONSTEXPR (do_g) {
      // torque
      real3 trqa = cross(phi1[1], phi1[2], phi1[3], di);
      trqa.x += phi1[9] * (qizz - qiyy) + 2 * (phi1[5] - phi1[6]) * qiyz +
         phi1[7] * qixz - phi1[8] * qixy;
      trqa.y += phi1[8] * (qixx - qizz) + 2 * (phi1[6] - phi1[4]) * qixz +
         phi1[9] * qixy - phi1[7] * qiyz;
      trqa.z += phi1[7] * (qiyy - qixx) + 2 * (phi1[4] - phi1[5]) * qixy +
         phi1[8] * qiyz - phi1[9] * qixz;
      real3 trqb = cross(phi2[1], phi2[2], phi2[3], dk);
      trqb.x += phi2[9] * (qkzz - qkyy) + 2 * (phi2[5] - phi2[6]) * qkyz +
         phi2[7] * qkxz - phi2[8] * qkxy;
      trqb.y += phi2[8] * (qkxx - qkzz) + 2 * (phi2[6] - phi2[4]) * qkxz +
         phi2[9] * qkxy - phi2[7] * qkyz;
      trqb.z += phi2[7] * (qkyy - qkxx) + 2 * (phi2[4] - phi2[5]) * qkxy +
         phi2[8] * qkyz - phi2[9] * qkxz;
      trq1 = trqa;
      trq2 = trqb;


      real3 trqau =
         cross(phi1d[0], phi1d[1], phi1d[2], (dscale * uip + pscale * uid));
      real3 trqbu =
         cross(phi2d[0], phi2d[1], phi2d[2], (dscale * ukp + pscale * ukd));


      // gradient
      real frc1z = phi1z[0] * ci + phi1z[1] * di.x + phi1z[2] * di.y +
         phi1z[3] * di.z + phi1z[4] * qixx + phi1z[5] * qiyy + phi1z[6] * qizz +
         phi1z[7] * qixy + phi1z[8] * qixz + phi1z[9] * qiyz;
      frc1z +=
         dot3(phi1dz[0], phi1dz[1], phi1dz[2], (dscale * uip + pscale * uid));
      frc.x = -invr1 * (trqa.y + trqb.y + trqau.y + trqbu.y);
      frc.y = invr1 * (trqa.x + trqb.x + trqau.x + trqbu.x);
      frc.z = frc1z;
   }


   // U-U
   {
      real coeu5 = uscale * sr5 * r;
      real coeu7 = uscale * sr7 * r2 * r;
      frc.x += coeu5 *
         (uid.x * ukp.z + uid.z * ukp.x + uip.x * ukd.z + uip.z * ukd.x);
      frc.y += coeu5 *
         (uid.y * ukp.z + uid.z * ukp.y + uip.y * ukd.z + uip.z * ukd.y);
      frc.z += coeu5 *
            (uid.x * ukp.x + uid.y * ukp.y + uip.x * ukd.x + uip.y * ukd.y) +
         (3 * coeu5 - coeu7) * (uid.z * ukp.z + uip.z * ukd.z);
   }


   if CONSTEXPR (do_g) {
      real3 glfrc;
      rotQI2GVector(rot, frc, glfrc);
      frc = f * glfrc;
      frcxi += frc.x;
      frcyi += frc.y;
      frczi += frc.z;
      frcxk -= frc.x;
      frcyk -= frc.y;
      frczk -= frc.z;
      real3 gltrq1;
      rotQI2GVector(rot, trq1, gltrq1);
      trqxi += f * gltrq1.x;
      trqyi += f * gltrq1.y;
      trqzi += f * gltrq1.z;
      real3 gltrq2;
      rotQI2GVector(rot, trq2, gltrq2);
      trqxk += f * gltrq2.x;
      trqyk += f * gltrq2.y;
      trqzk += f * gltrq2.z;
   }
   if CONSTEXPR (do_v) {
      voxx = -dR.x * frc.x;
      voxy = -0.5f * (dR.y * frc.x + dR.x * frc.y);
      voxz = -0.5f * (dR.z * frc.x + dR.x * frc.z);
      voyy = -dR.y * frc.y;
      voyz = -0.5f * (dR.z * frc.y + dR.y * frc.z);
      vozz = -dR.z * frc.z;
   }
}


// ck.py Version 2.0.1


template <class Ver, class ETYP>
__global__
void emplar_cu1c(TINKER_IMAGE_PARAMS, energy_buffer restrict ebuf,
                 virial_buffer restrict vbuf, grad_prec* restrict gx,
                 grad_prec* restrict gy, grad_prec* restrict gz, real off,
                 real* restrict trqx, real* restrict trqy, real* restrict trqz,
                 const real (*restrict rpole)[10],
                 const real (*restrict uind)[3], const real (*restrict uinp)[3],
                 const real* restrict thole, const real* restrict pdamp, real f,
                 real aewald, int nexclude, const int (*restrict exclude)[2],
                 const real (*restrict exclude_scale)[4],
                 const real* restrict x, const real* restrict y,
                 const real* restrict z)
{
   constexpr bool do_e = Ver::e;
   constexpr bool do_g = Ver::g;
   constexpr bool do_v = Ver::v;
   static_assert(!Ver::a, "");
   const int ithread = threadIdx.x + blockIdx.x * blockDim.x;


   using ebuf_prec = energy_buffer_traits::type;
   ebuf_prec ebuftl;
   if CONSTEXPR (do_e) {
      ebuftl = 0;
   }
   using vbuf_prec = virial_buffer_traits::type;
   vbuf_prec vbuftlxx, vbuftlyx, vbuftlzx, vbuftlyy, vbuftlzy, vbuftlzz;
   if CONSTEXPR (do_v) {
      vbuftlxx = 0;
      vbuftlyx = 0;
      vbuftlzx = 0;
      vbuftlyy = 0;
      vbuftlzy = 0;
      vbuftlzz = 0;
   }
   __shared__ real xi[BLOCK_DIM];
   __shared__ real yi[BLOCK_DIM];
   __shared__ real zi[BLOCK_DIM];
   real xk;
   real yk;
   real zk;
   __shared__ real frcxi[BLOCK_DIM];
   __shared__ real frcyi[BLOCK_DIM];
   __shared__ real frczi[BLOCK_DIM];
   __shared__ real trqxi[BLOCK_DIM];
   __shared__ real trqyi[BLOCK_DIM];
   __shared__ real trqzi[BLOCK_DIM];
   real frcxk;
   real frcyk;
   real frczk;
   real trqxk;
   real trqyk;
   real trqzk;
   __shared__ real ci[BLOCK_DIM];
   __shared__ real dix[BLOCK_DIM];
   __shared__ real diy[BLOCK_DIM];
   __shared__ real diz[BLOCK_DIM];
   __shared__ real qixx[BLOCK_DIM];
   __shared__ real qixy[BLOCK_DIM];
   __shared__ real qixz[BLOCK_DIM];
   __shared__ real qiyy[BLOCK_DIM];
   __shared__ real qiyz[BLOCK_DIM];
   __shared__ real qizz[BLOCK_DIM];
   __shared__ real uidx[BLOCK_DIM];
   __shared__ real uidy[BLOCK_DIM];
   __shared__ real uidz[BLOCK_DIM];
   __shared__ real uipx[BLOCK_DIM];
   __shared__ real uipy[BLOCK_DIM];
   __shared__ real uipz[BLOCK_DIM];
   __shared__ real pdi[BLOCK_DIM];
   __shared__ real pti[BLOCK_DIM];
   real ck;
   real dkx;
   real dky;
   real dkz;
   real qkxx;
   real qkxy;
   real qkxz;
   real qkyy;
   real qkyz;
   real qkzz;
   real ukdx;
   real ukdy;
   real ukdz;
   real ukpx;
   real ukpy;
   real ukpz;
   real pdk;
   real ptk;


   for (int ii = ithread; ii < nexclude; ii += blockDim.x * gridDim.x) {
      const int klane = threadIdx.x;
      if CONSTEXPR (do_g) {
         frcxi[threadIdx.x] = 0;
         frcyi[threadIdx.x] = 0;
         frczi[threadIdx.x] = 0;
         trqxi[threadIdx.x] = 0;
         trqyi[threadIdx.x] = 0;
         trqzi[threadIdx.x] = 0;
         frcxk = 0;
         frcyk = 0;
         frczk = 0;
         trqxk = 0;
         trqyk = 0;
         trqzk = 0;
      }


      int i = exclude[ii][0];
      int k = exclude[ii][1];
      real scalea = exclude_scale[ii][0];
      real scaleb = exclude_scale[ii][1];
      real scalec = exclude_scale[ii][2];
      real scaled = exclude_scale[ii][3];


      xi[klane] = x[i];
      yi[klane] = y[i];
      zi[klane] = z[i];
      xk = x[k];
      yk = y[k];
      zk = z[k];
      ci[klane] = rpole[i][mpl_pme_0];
      dix[klane] = rpole[i][mpl_pme_x];
      diy[klane] = rpole[i][mpl_pme_y];
      diz[klane] = rpole[i][mpl_pme_z];
      qixx[klane] = rpole[i][mpl_pme_xx];
      qixy[klane] = rpole[i][mpl_pme_xy];
      qixz[klane] = rpole[i][mpl_pme_xz];
      qiyy[klane] = rpole[i][mpl_pme_yy];
      qiyz[klane] = rpole[i][mpl_pme_yz];
      qizz[klane] = rpole[i][mpl_pme_zz];
      uidx[klane] = uind[i][0];
      uidy[klane] = uind[i][1];
      uidz[klane] = uind[i][2];
      uipx[klane] = uinp[i][0];
      uipy[klane] = uinp[i][1];
      uipz[klane] = uinp[i][2];
      pdi[klane] = pdamp[i];
      pti[klane] = thole[i];
      ck = rpole[k][mpl_pme_0];
      dkx = rpole[k][mpl_pme_x];
      dky = rpole[k][mpl_pme_y];
      dkz = rpole[k][mpl_pme_z];
      qkxx = rpole[k][mpl_pme_xx];
      qkxy = rpole[k][mpl_pme_xy];
      qkxz = rpole[k][mpl_pme_xz];
      qkyy = rpole[k][mpl_pme_yy];
      qkyz = rpole[k][mpl_pme_yz];
      qkzz = rpole[k][mpl_pme_zz];
      ukdx = uind[k][0];
      ukdy = uind[k][1];
      ukdz = uind[k][2];
      ukpx = uinp[k][0];
      ukpy = uinp[k][1];
      ukpz = uinp[k][2];
      pdk = pdamp[k];
      ptk = thole[k];


      constexpr bool incl = true;
      real xr = xk - xi[klane];
      real yr = yk - yi[klane];
      real zr = zk - zi[klane];
      real r2 = image2(xr, yr, zr);
      if (r2 <= off * off and incl) {
         real e1, vxx1, vyx1, vzx1, vyy1, vzy1, vzz1;
         pair_mplar<Ver, NON_EWALD>(
            r2, make_real3(xr, yr, zr), scalea - 1, scaleb - 1, scalec - 1,
            scaled - 1, ci[klane],
            make_real3(dix[klane], diy[klane], diz[klane]), qixx[klane],
            qixy[klane], qixz[klane], qiyy[klane], qiyz[klane], qizz[klane],
            make_real3(uidx[klane], uidy[klane], uidz[klane]),
            make_real3(uipx[klane], uipy[klane], uipz[klane]), pdi[klane],
            pti[klane], ck, make_real3(dkx, dky, dkz), qkxx, qkxy, qkxz, qkyy,
            qkyz, qkzz, make_real3(ukdx, ukdy, ukdz),
            make_real3(ukpx, ukpy, ukpz), pdk, ptk, f, aewald, frcxi[klane],
            frcyi[klane], frczi[klane], frcxk, frcyk, frczk, trqxi[klane],
            trqyi[klane], trqzi[klane], trqxk, trqyk, trqzk, e1, vxx1, vyx1,
            vzx1, vyy1, vzy1, vzz1);
         if CONSTEXPR (do_e) {
            ebuftl += cvt_to<ebuf_prec>(e1);
         }
         if CONSTEXPR (do_v) {
            vbuftlxx += cvt_to<vbuf_prec>(vxx1);
            vbuftlyx += cvt_to<vbuf_prec>(vyx1);
            vbuftlzx += cvt_to<vbuf_prec>(vzx1);
            vbuftlyy += cvt_to<vbuf_prec>(vyy1);
            vbuftlzy += cvt_to<vbuf_prec>(vzy1);
            vbuftlzz += cvt_to<vbuf_prec>(vzz1);
         }
      } // end if (include)


      if CONSTEXPR (do_g) {
         atomic_add(frcxi[threadIdx.x], gx, i);
         atomic_add(frcyi[threadIdx.x], gy, i);
         atomic_add(frczi[threadIdx.x], gz, i);
         atomic_add(trqxi[threadIdx.x], trqx, i);
         atomic_add(trqyi[threadIdx.x], trqy, i);
         atomic_add(trqzi[threadIdx.x], trqz, i);
         atomic_add(frcxk, gx, k);
         atomic_add(frcyk, gy, k);
         atomic_add(frczk, gz, k);
         atomic_add(trqxk, trqx, k);
         atomic_add(trqyk, trqy, k);
         atomic_add(trqzk, trqz, k);
      }
   }


   if CONSTEXPR (do_e) {
      atomic_add(ebuftl, ebuf, ithread);
   }
   if CONSTEXPR (do_v) {
      atomic_add(vbuftlxx, vbuftlyx, vbuftlzx, vbuftlyy, vbuftlzy, vbuftlzz,
                 vbuf, ithread);
   }
}


template <class Ver, class ETYP>
__global__
void emplar_cu1b(TINKER_IMAGE_PARAMS, energy_buffer restrict ebuf,
                 virial_buffer restrict vbuf, grad_prec* restrict gx,
                 grad_prec* restrict gy, grad_prec* restrict gz, real off,
                 real* restrict trqx, real* restrict trqy, real* restrict trqz,
                 const real (*restrict rpole)[10],
                 const real (*restrict uind)[3], const real (*restrict uinp)[3],
                 const real* restrict thole, const real* restrict pdamp, real f,
                 real aewald, const Spatial::SortedAtom* restrict sorted, int n,
                 int nakpl, const int* restrict iakpl)
{
   constexpr bool do_e = Ver::e;
   constexpr bool do_g = Ver::g;
   constexpr bool do_v = Ver::v;
   static_assert(!Ver::a, "");
   const int ithread = threadIdx.x + blockIdx.x * blockDim.x;
   const int iwarp = ithread / WARP_SIZE;
   const int nwarp = blockDim.x * gridDim.x / WARP_SIZE;
   const int ilane = threadIdx.x & (WARP_SIZE - 1);


   using ebuf_prec = energy_buffer_traits::type;
   ebuf_prec ebuftl;
   if CONSTEXPR (do_e) {
      ebuftl = 0;
   }
   using vbuf_prec = virial_buffer_traits::type;
   vbuf_prec vbuftlxx, vbuftlyx, vbuftlzx, vbuftlyy, vbuftlzy, vbuftlzz;
   if CONSTEXPR (do_v) {
      vbuftlxx = 0;
      vbuftlyx = 0;
      vbuftlzx = 0;
      vbuftlyy = 0;
      vbuftlzy = 0;
      vbuftlzz = 0;
   }
   __shared__ real xi[BLOCK_DIM];
   __shared__ real yi[BLOCK_DIM];
   __shared__ real zi[BLOCK_DIM];
   real xk;
   real yk;
   real zk;
   __shared__ real frcxi[BLOCK_DIM];
   __shared__ real frcyi[BLOCK_DIM];
   __shared__ real frczi[BLOCK_DIM];
   __shared__ real trqxi[BLOCK_DIM];
   __shared__ real trqyi[BLOCK_DIM];
   __shared__ real trqzi[BLOCK_DIM];
   real frcxk;
   real frcyk;
   real frczk;
   real trqxk;
   real trqyk;
   real trqzk;
   __shared__ real ci[BLOCK_DIM];
   __shared__ real dix[BLOCK_DIM];
   __shared__ real diy[BLOCK_DIM];
   __shared__ real diz[BLOCK_DIM];
   __shared__ real qixx[BLOCK_DIM];
   __shared__ real qixy[BLOCK_DIM];
   __shared__ real qixz[BLOCK_DIM];
   __shared__ real qiyy[BLOCK_DIM];
   __shared__ real qiyz[BLOCK_DIM];
   __shared__ real qizz[BLOCK_DIM];
   __shared__ real uidx[BLOCK_DIM];
   __shared__ real uidy[BLOCK_DIM];
   __shared__ real uidz[BLOCK_DIM];
   __shared__ real uipx[BLOCK_DIM];
   __shared__ real uipy[BLOCK_DIM];
   __shared__ real uipz[BLOCK_DIM];
   __shared__ real pdi[BLOCK_DIM];
   __shared__ real pti[BLOCK_DIM];
   real ck;
   real dkx;
   real dky;
   real dkz;
   real qkxx;
   real qkxy;
   real qkxz;
   real qkyy;
   real qkyz;
   real qkzz;
   real ukdx;
   real ukdy;
   real ukdz;
   real ukpx;
   real ukpy;
   real ukpz;
   real pdk;
   real ptk;


   for (int iw = iwarp; iw < nakpl; iw += nwarp) {
      if CONSTEXPR (do_g) {
         frcxi[threadIdx.x] = 0;
         frcyi[threadIdx.x] = 0;
         frczi[threadIdx.x] = 0;
         trqxi[threadIdx.x] = 0;
         trqyi[threadIdx.x] = 0;
         trqzi[threadIdx.x] = 0;
         frcxk = 0;
         frcyk = 0;
         frczk = 0;
         trqxk = 0;
         trqyk = 0;
         trqzk = 0;
      }


      int tri, tx, ty;
      tri = iakpl[iw];
      tri_to_xy(tri, tx, ty);


      int iid = ty * WARP_SIZE + ilane;
      int atomi = min(iid, n - 1);
      int i = sorted[atomi].unsorted;
      int kid = tx * WARP_SIZE + ilane;
      int atomk = min(kid, n - 1);
      int k = sorted[atomk].unsorted;
      xi[threadIdx.x] = sorted[atomi].x;
      yi[threadIdx.x] = sorted[atomi].y;
      zi[threadIdx.x] = sorted[atomi].z;
      xk = sorted[atomk].x;
      yk = sorted[atomk].y;
      zk = sorted[atomk].z;


      ci[threadIdx.x] = rpole[i][mpl_pme_0];
      dix[threadIdx.x] = rpole[i][mpl_pme_x];
      diy[threadIdx.x] = rpole[i][mpl_pme_y];
      diz[threadIdx.x] = rpole[i][mpl_pme_z];
      qixx[threadIdx.x] = rpole[i][mpl_pme_xx];
      qixy[threadIdx.x] = rpole[i][mpl_pme_xy];
      qixz[threadIdx.x] = rpole[i][mpl_pme_xz];
      qiyy[threadIdx.x] = rpole[i][mpl_pme_yy];
      qiyz[threadIdx.x] = rpole[i][mpl_pme_yz];
      qizz[threadIdx.x] = rpole[i][mpl_pme_zz];
      uidx[threadIdx.x] = uind[i][0];
      uidy[threadIdx.x] = uind[i][1];
      uidz[threadIdx.x] = uind[i][2];
      uipx[threadIdx.x] = uinp[i][0];
      uipy[threadIdx.x] = uinp[i][1];
      uipz[threadIdx.x] = uinp[i][2];
      pdi[threadIdx.x] = pdamp[i];
      pti[threadIdx.x] = thole[i];
      ck = rpole[k][mpl_pme_0];
      dkx = rpole[k][mpl_pme_x];
      dky = rpole[k][mpl_pme_y];
      dkz = rpole[k][mpl_pme_z];
      qkxx = rpole[k][mpl_pme_xx];
      qkxy = rpole[k][mpl_pme_xy];
      qkxz = rpole[k][mpl_pme_xz];
      qkyy = rpole[k][mpl_pme_yy];
      qkyz = rpole[k][mpl_pme_yz];
      qkzz = rpole[k][mpl_pme_zz];
      ukdx = uind[k][0];
      ukdy = uind[k][1];
      ukdz = uind[k][2];
      ukpx = uinp[k][0];
      ukpy = uinp[k][1];
      ukpz = uinp[k][2];
      pdk = pdamp[k];
      ptk = thole[k];


      for (int j = 0; j < WARP_SIZE; ++j) {
         int srclane = (ilane + j) & (WARP_SIZE - 1);
         int klane = srclane + threadIdx.x - ilane;
         bool incl = iid < kid and kid < n;
         real xr = xk - xi[klane];
         real yr = yk - yi[klane];
         real zr = zk - zi[klane];
         real r2 = image2(xr, yr, zr);
         if (r2 <= off * off and incl) {
            real e, vxx, vyx, vzx, vyy, vzy, vzz;
            pair_mplar<Ver, ETYP>(
               r2, make_real3(xr, yr, zr), 1, 1, 1, 1, ci[klane],
               make_real3(dix[klane], diy[klane], diz[klane]), qixx[klane],
               qixy[klane], qixz[klane], qiyy[klane], qiyz[klane], qizz[klane],
               make_real3(uidx[klane], uidy[klane], uidz[klane]),
               make_real3(uipx[klane], uipy[klane], uipz[klane]), pdi[klane],
               pti[klane], ck, make_real3(dkx, dky, dkz), qkxx, qkxy, qkxz,
               qkyy, qkyz, qkzz, make_real3(ukdx, ukdy, ukdz),
               make_real3(ukpx, ukpy, ukpz), pdk, ptk, f, aewald, frcxi[klane],
               frcyi[klane], frczi[klane], frcxk, frcyk, frczk, trqxi[klane],
               trqyi[klane], trqzi[klane], trqxk, trqyk, trqzk, e, vxx, vyx,
               vzx, vyy, vzy, vzz);
            if CONSTEXPR (do_e) {
               ebuftl += cvt_to<ebuf_prec>(e);
            }
            if CONSTEXPR (do_v) {
               vbuftlxx += cvt_to<vbuf_prec>(vxx);
               vbuftlyx += cvt_to<vbuf_prec>(vyx);
               vbuftlzx += cvt_to<vbuf_prec>(vzx);
               vbuftlyy += cvt_to<vbuf_prec>(vyy);
               vbuftlzy += cvt_to<vbuf_prec>(vzy);
               vbuftlzz += cvt_to<vbuf_prec>(vzz);
            }
         } // end if (include)


         iid = __shfl_sync(ALL_LANES, iid, ilane + 1);
      }


      if CONSTEXPR (do_g) {
         atomic_add(frcxi[threadIdx.x], gx, i);
         atomic_add(frcyi[threadIdx.x], gy, i);
         atomic_add(frczi[threadIdx.x], gz, i);
         atomic_add(trqxi[threadIdx.x], trqx, i);
         atomic_add(trqyi[threadIdx.x], trqy, i);
         atomic_add(trqzi[threadIdx.x], trqz, i);
         atomic_add(frcxk, gx, k);
         atomic_add(frcyk, gy, k);
         atomic_add(frczk, gz, k);
         atomic_add(trqxk, trqx, k);
         atomic_add(trqyk, trqy, k);
         atomic_add(trqzk, trqz, k);
      }
   }


   if CONSTEXPR (do_e) {
      atomic_add(ebuftl, ebuf, ithread);
   }
   if CONSTEXPR (do_v) {
      atomic_add(vbuftlxx, vbuftlyx, vbuftlzx, vbuftlyy, vbuftlzy, vbuftlzz,
                 vbuf, ithread);
   }
}


template <class Ver, class ETYP>
__global__
void emplar_cu1a(TINKER_IMAGE_PARAMS, energy_buffer restrict ebuf,
                 virial_buffer restrict vbuf, grad_prec* restrict gx,
                 grad_prec* restrict gy, grad_prec* restrict gz, real off,
                 real* restrict trqx, real* restrict trqy, real* restrict trqz,
                 const real (*restrict rpole)[10],
                 const real (*restrict uind)[3], const real (*restrict uinp)[3],
                 const real* restrict thole, const real* restrict pdamp, real f,
                 real aewald, const Spatial::SortedAtom* restrict sorted,
                 int niak, const int* restrict iak, const int* restrict lst)
{
   constexpr bool do_e = Ver::e;
   constexpr bool do_g = Ver::g;
   constexpr bool do_v = Ver::v;
   static_assert(!Ver::a, "");
   const int ithread = threadIdx.x + blockIdx.x * blockDim.x;
   const int iwarp = ithread / WARP_SIZE;
   const int nwarp = blockDim.x * gridDim.x / WARP_SIZE;
   const int ilane = threadIdx.x & (WARP_SIZE - 1);


   using ebuf_prec = energy_buffer_traits::type;
   ebuf_prec ebuftl;
   if CONSTEXPR (do_e) {
      ebuftl = 0;
   }
   using vbuf_prec = virial_buffer_traits::type;
   vbuf_prec vbuftlxx, vbuftlyx, vbuftlzx, vbuftlyy, vbuftlzy, vbuftlzz;
   if CONSTEXPR (do_v) {
      vbuftlxx = 0;
      vbuftlyx = 0;
      vbuftlzx = 0;
      vbuftlyy = 0;
      vbuftlzy = 0;
      vbuftlzz = 0;
   }
   __shared__ real xi[BLOCK_DIM];
   __shared__ real yi[BLOCK_DIM];
   __shared__ real zi[BLOCK_DIM];
   real xk;
   real yk;
   real zk;
   __shared__ real frcxi[BLOCK_DIM];
   __shared__ real frcyi[BLOCK_DIM];
   __shared__ real frczi[BLOCK_DIM];
   __shared__ real trqxi[BLOCK_DIM];
   __shared__ real trqyi[BLOCK_DIM];
   __shared__ real trqzi[BLOCK_DIM];
   real frcxk;
   real frcyk;
   real frczk;
   real trqxk;
   real trqyk;
   real trqzk;
   __shared__ real ci[BLOCK_DIM];
   __shared__ real dix[BLOCK_DIM];
   __shared__ real diy[BLOCK_DIM];
   __shared__ real diz[BLOCK_DIM];
   __shared__ real qixx[BLOCK_DIM];
   __shared__ real qixy[BLOCK_DIM];
   __shared__ real qixz[BLOCK_DIM];
   __shared__ real qiyy[BLOCK_DIM];
   __shared__ real qiyz[BLOCK_DIM];
   __shared__ real qizz[BLOCK_DIM];
   __shared__ real uidx[BLOCK_DIM];
   __shared__ real uidy[BLOCK_DIM];
   __shared__ real uidz[BLOCK_DIM];
   __shared__ real uipx[BLOCK_DIM];
   __shared__ real uipy[BLOCK_DIM];
   __shared__ real uipz[BLOCK_DIM];
   __shared__ real pdi[BLOCK_DIM];
   __shared__ real pti[BLOCK_DIM];
   real ck;
   real dkx;
   real dky;
   real dkz;
   real qkxx;
   real qkxy;
   real qkxz;
   real qkyy;
   real qkyz;
   real qkzz;
   real ukdx;
   real ukdy;
   real ukdz;
   real ukpx;
   real ukpy;
   real ukpz;
   real pdk;
   real ptk;


   for (int iw = iwarp; iw < niak; iw += nwarp) {
      if CONSTEXPR (do_g) {
         frcxi[threadIdx.x] = 0;
         frcyi[threadIdx.x] = 0;
         frczi[threadIdx.x] = 0;
         trqxi[threadIdx.x] = 0;
         trqyi[threadIdx.x] = 0;
         trqzi[threadIdx.x] = 0;
         frcxk = 0;
         frcyk = 0;
         frczk = 0;
         trqxk = 0;
         trqyk = 0;
         trqzk = 0;
      }


      int ty = iak[iw];
      int atomi = ty * WARP_SIZE + ilane;
      int i = sorted[atomi].unsorted;
      int atomk = lst[iw * WARP_SIZE + ilane];
      int k = sorted[atomk].unsorted;
      xi[threadIdx.x] = sorted[atomi].x;
      yi[threadIdx.x] = sorted[atomi].y;
      zi[threadIdx.x] = sorted[atomi].z;
      xk = sorted[atomk].x;
      yk = sorted[atomk].y;
      zk = sorted[atomk].z;


      ci[threadIdx.x] = rpole[i][mpl_pme_0];
      dix[threadIdx.x] = rpole[i][mpl_pme_x];
      diy[threadIdx.x] = rpole[i][mpl_pme_y];
      diz[threadIdx.x] = rpole[i][mpl_pme_z];
      qixx[threadIdx.x] = rpole[i][mpl_pme_xx];
      qixy[threadIdx.x] = rpole[i][mpl_pme_xy];
      qixz[threadIdx.x] = rpole[i][mpl_pme_xz];
      qiyy[threadIdx.x] = rpole[i][mpl_pme_yy];
      qiyz[threadIdx.x] = rpole[i][mpl_pme_yz];
      qizz[threadIdx.x] = rpole[i][mpl_pme_zz];
      uidx[threadIdx.x] = uind[i][0];
      uidy[threadIdx.x] = uind[i][1];
      uidz[threadIdx.x] = uind[i][2];
      uipx[threadIdx.x] = uinp[i][0];
      uipy[threadIdx.x] = uinp[i][1];
      uipz[threadIdx.x] = uinp[i][2];
      pdi[threadIdx.x] = pdamp[i];
      pti[threadIdx.x] = thole[i];
      ck = rpole[k][mpl_pme_0];
      dkx = rpole[k][mpl_pme_x];
      dky = rpole[k][mpl_pme_y];
      dkz = rpole[k][mpl_pme_z];
      qkxx = rpole[k][mpl_pme_xx];
      qkxy = rpole[k][mpl_pme_xy];
      qkxz = rpole[k][mpl_pme_xz];
      qkyy = rpole[k][mpl_pme_yy];
      qkyz = rpole[k][mpl_pme_yz];
      qkzz = rpole[k][mpl_pme_zz];
      ukdx = uind[k][0];
      ukdy = uind[k][1];
      ukdz = uind[k][2];
      ukpx = uinp[k][0];
      ukpy = uinp[k][1];
      ukpz = uinp[k][2];
      pdk = pdamp[k];
      ptk = thole[k];


      for (int j = 0; j < WARP_SIZE; ++j) {
         int srclane = (ilane + j) & (WARP_SIZE - 1);
         int klane = srclane + threadIdx.x - ilane;
         bool incl = atomk > 0;
         real xr = xk - xi[klane];
         real yr = yk - yi[klane];
         real zr = zk - zi[klane];
         real r2 = image2(xr, yr, zr);
         if (r2 <= off * off and incl) {
            real e, vxx, vyx, vzx, vyy, vzy, vzz;
            pair_mplar<Ver, ETYP>(
               r2, make_real3(xr, yr, zr), 1, 1, 1, 1, ci[klane],
               make_real3(dix[klane], diy[klane], diz[klane]), qixx[klane],
               qixy[klane], qixz[klane], qiyy[klane], qiyz[klane], qizz[klane],
               make_real3(uidx[klane], uidy[klane], uidz[klane]),
               make_real3(uipx[klane], uipy[klane], uipz[klane]), pdi[klane],
               pti[klane], ck, make_real3(dkx, dky, dkz), qkxx, qkxy, qkxz,
               qkyy, qkyz, qkzz, make_real3(ukdx, ukdy, ukdz),
               make_real3(ukpx, ukpy, ukpz), pdk, ptk, f, aewald, frcxi[klane],
               frcyi[klane], frczi[klane], frcxk, frcyk, frczk, trqxi[klane],
               trqyi[klane], trqzi[klane], trqxk, trqyk, trqzk, e, vxx, vyx,
               vzx, vyy, vzy, vzz);
            if CONSTEXPR (do_e) {
               ebuftl += cvt_to<ebuf_prec>(e);
            }
            if CONSTEXPR (do_v) {
               vbuftlxx += cvt_to<vbuf_prec>(vxx);
               vbuftlyx += cvt_to<vbuf_prec>(vyx);
               vbuftlzx += cvt_to<vbuf_prec>(vzx);
               vbuftlyy += cvt_to<vbuf_prec>(vyy);
               vbuftlzy += cvt_to<vbuf_prec>(vzy);
               vbuftlzz += cvt_to<vbuf_prec>(vzz);
            }
         } // end if (include)
      }


      if CONSTEXPR (do_g) {
         atomic_add(frcxi[threadIdx.x], gx, i);
         atomic_add(frcyi[threadIdx.x], gy, i);
         atomic_add(frczi[threadIdx.x], gz, i);
         atomic_add(trqxi[threadIdx.x], trqx, i);
         atomic_add(trqyi[threadIdx.x], trqy, i);
         atomic_add(trqzi[threadIdx.x], trqz, i);
         atomic_add(frcxk, gx, k);
         atomic_add(frcyk, gy, k);
         atomic_add(frczk, gz, k);
         atomic_add(trqxk, trqx, k);
         atomic_add(trqyk, trqy, k);
         atomic_add(trqzk, trqz, k);
      }
   }


   if CONSTEXPR (do_e) {
      atomic_add(ebuftl, ebuf, ithread);
   }
   if CONSTEXPR (do_v) {
      atomic_add(vbuftlxx, vbuftlyx, vbuftlzx, vbuftlyy, vbuftlzy, vbuftlzz,
                 vbuf, ithread);
   }
}


template <class Ver, class ETYP>
void emplar_cu(const real (*uind)[3], const real (*uinp)[3])
{
   const auto& st = *mspatial_v2_unit;
   real off;
   if CONSTEXPR (eq<ETYP, EWALD>())
      off = switch_off(switch_ewald);
   else
      off = switch_off(switch_mpole);


   const real f = electric / dielec;
   real aewald = 0;
   if CONSTEXPR (eq<ETYP, EWALD>()) {
      assert(epme_unit == ppme_unit);
      PMEUnit pu = epme_unit;
      aewald = pu->aewald;


      if CONSTEXPR (Ver::e) {
         auto ker0 = empole_self_cu<Ver::a>;
         launch_k1b(g::s0, n, ker0, //
                    nullptr, em, rpole, n, f, aewald);
      }
   }
   int ngrid = get_grid_size(BLOCK_DIM);
   auto kera = emplar_cu1a<Ver, ETYP>;
   kera<<<ngrid, BLOCK_DIM, 0, g::s0>>>(
      TINKER_IMAGE_ARGS, em, vir_em, demx, demy, demz, off, trqx, trqy, trqz,
      rpole, uind, uinp, thole, pdamp, f, aewald, //
      st.sorted, st.niak, st.iak, st.lst);
   auto kerb = emplar_cu1b<Ver, ETYP>;
   kerb<<<ngrid, BLOCK_DIM, 0, g::s0>>>(
      TINKER_IMAGE_ARGS, em, vir_em, demx, demy, demz, off, trqx, trqy, trqz,
      rpole, uind, uinp, thole, pdamp, f, aewald, //
      st.sorted, st.n, st.nakpl, st.iakpl);
   auto kerc = emplar_cu1c<Ver, ETYP>;
   kerc<<<ngrid, BLOCK_DIM, 0, g::s0>>>(
      TINKER_IMAGE_ARGS, em, vir_em, demx, demy, demz, off, trqx, trqy, trqz,
      rpole, uind, uinp, thole, pdamp, f, aewald, //
      nmdpuexclude, mdpuexclude, mdpuexclude_scale, st.x, st.y, st.z);
}


template <class Ver>
void emplar_ewald_cu()
{
   // induce
   induce(uind, uinp);

   // empole real self; epolar real without epolar energy
   emplar_cu<Ver, EWALD>(uind, uinp);
   // empole recip
   empole_ewald_recip(Ver::value);
   // epolar recip self; must toggle off the calc::energy flag
   epolar_ewald_recip_self(Ver::value & ~calc::energy);

   // epolar energy
   if CONSTEXPR (Ver::e)
      epolar0_dotprod(uind, udirp);
}


template <class Ver>
void emplar_nonewald_cu()
{
   // induce
   induce(uind, uinp);

   // empole and epolar
   emplar_cu<Ver, NON_EWALD>(uind, uinp);
   if CONSTEXPR (Ver::e)
      epolar0_dotprod(uind, udirp);
}


void emplar_cu(int vers)
{
   if (use_ewald()) {
      if (vers == calc::v0)
         emplar_ewald_cu<calc::V0>();
      else if (vers == calc::v1)
         emplar_ewald_cu<calc::V1>();
      // else if (vers == calc::v3)
      //    emplar_ewald_cu<calc::V3>();
      else if (vers == calc::v4)
         emplar_ewald_cu<calc::V4>();
      else if (vers == calc::v5)
         emplar_ewald_cu<calc::V5>();
      else if (vers == calc::v6)
         emplar_ewald_cu<calc::V6>();
   } else {
      if (vers == calc::v0)
         emplar_nonewald_cu<calc::V0>();
      else if (vers == calc::v1)
         emplar_nonewald_cu<calc::V1>();
      // else if (vers == calc::v3)
      //    emplar_nonewald_cu<calc::V3>();
      else if (vers == calc::v4)
         emplar_nonewald_cu<calc::V4>();
      else if (vers == calc::v5)
         emplar_nonewald_cu<calc::V5>();
      else if (vers == calc::v6)
         emplar_nonewald_cu<calc::V6>();
   }
}
}
